#include <hip/hip_runtime.h>

#include <iomanip>
#include <iostream>
#include <malloc.h>
#include <stdio.h>

using namespace std;

#define CUDA_CHECK_RETURN(value)                                               \
  {                                                                            \
    hipError_t _m_cudaStat = value;                                           \
    if (_m_cudaStat != hipSuccess) {                                          \
      fprintf(stderr, "Error %s at line %d in file %s\n",                      \
              hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);            \
      exit(1);                                                                 \
    }                                                                          \
  }

void Output(float *a, int N) {
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++)
      fprintf(stdout, "%g\t", a[j + i * N]);
    fprintf(stdout, "\n");
  }
  fprintf(stdout, "\n\n\n");
}



__global__ void gTrans(float* matrix ,float* matrixT) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  int N = blockDim.x*gridDim.x;

  matrixT[j+i*N] = matrix[i+j*N];

}


__global__ void gInit(float* matrix) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  int N = blockDim.x*gridDim.x;

  matrix[i+j*N] = (float)(i+j*N);

}

__global__ void gInit1(float* matrix) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  int N = blockDim.y*gridDim.y;

  matrix[j+i*N] = (float)(j+i*N);

}



int main(int argc,char* argv[]) {
  float elapsedTime;
  int N = 8192;
  int dimB = 32;
  int dimG = N/dimB;
  float *dev_matrix,*dev_matrixT,*dev_matrix1,*matrix,*matrixT,*matrix1;
  hipEvent_t start, stop;
  hipEventCreate(&start); // инициализация
  hipEventCreate(&stop); 

  matrix = (float *)calloc(N*N, sizeof(float));
  matrix1 = (float *)calloc(N*N, sizeof(float));
  matrixT = (float *)calloc(N*N, sizeof(float));
  CUDA_CHECK_RETURN(hipMalloc((void **)&dev_matrix, N*N * sizeof(float)));
  CUDA_CHECK_RETURN(hipMalloc((void **)&dev_matrix1, N*N * sizeof(float)));
  CUDA_CHECK_RETURN(hipMalloc((void **)&dev_matrixT, N*N * sizeof(float)));

  gInit<<<dim3(dimG,dimG),dim3(dimB,dimB)>>>(dev_matrix);
  hipDeviceSynchronize();
  CUDA_CHECK_RETURN(hipMemcpy(matrix, dev_matrix, N * N * sizeof(float), hipMemcpyDeviceToHost));
  
  gInit1<<<dim3(dimG,dimG),dim3(dimB,dimB)>>>(dev_matrix1);
  hipDeviceSynchronize();
  CUDA_CHECK_RETURN(hipMemcpy(matrix1, dev_matrix1, N * N * sizeof(float), hipMemcpyDeviceToHost));

  hipEventRecord(start, 0); // привязка (регистрация) события start
  gTrans<<<dim3(dimG,dimG),dim3(dimB,dimB)>>>(dev_matrix,dev_matrixT);
  hipEventRecord(stop, 0); // привязка события stop
  hipEventSynchronize(stop); // синхронизация по событию
  CUDA_CHECK_RETURN(hipMemcpy(matrixT, dev_matrixT, N * N * sizeof(float), hipMemcpyDeviceToHost));
  CUDA_CHECK_RETURN(hipGetLastError());
  hipEventElapsedTime(&elapsedTime, start,stop);
  printf("gTranspose took %gms\n", elapsedTime);

  hipEventDestroy(start); // освобождение
  hipEventDestroy(stop);  // памяти
  free(matrix);
  free(matrixT);
  hipFree(dev_matrix);
  hipFree(dev_matrixT);
  return 0;
}
    
