#include <hip/hip_runtime.h>

#include <iomanip>
#include <iostream>
#include <malloc.h>
#include <stdio.h>

using namespace std;

#define CUDA_CHECK_RETURN(value)                                               \
  {                                                                            \
    hipError_t _m_cudaStat = value;                                           \
    if (_m_cudaStat != hipSuccess) {                                          \
      fprintf(stderr, "Error %s at line %d in file %s\n",                      \
              hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);            \
      exit(1);                                                                 \
    }                                                                          \
  }

__global__ void add(int*a, int *b, int *c, int N) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
  if(i >= N) return;
  c[i] = a[i] + b[i];
}

void init(int*a, int *b, int *c, int N)
{
    for(int k = 0; k < N; k++)
    {
      a[k] = k;
      b[k] = k;
      c[k] = 0;
    }
}

int main(int argc,char* argv[]) 
{ 
    float elapsedTime;
    int N = 0;
    int *dev_a, *dev_b, *dev_c, *c, *a, *b;

    if(argc == 3)
    {
         int N = atoi(argv[1]);
         int th = atoi(argv[2]);
         hipEvent_t start, stop;          
         hipEventCreate(&start); 
         hipEventCreate(&stop);

         a = (int*)calloc(N, sizeof(int));
         b = (int*)calloc(N, sizeof(int));
         c = (int*)calloc(N, sizeof(int));
         init(a, b, c, N);
         CUDA_CHECK_RETURN(hipMalloc((void **)&dev_a, N * sizeof(int)));
         CUDA_CHECK_RETURN(hipMalloc((void **)&dev_b, N * sizeof(int)));
         CUDA_CHECK_RETURN(hipMalloc((void **)&dev_c, N * sizeof(int)));

         hipEventRecord(start, 0); 
         add<<<N / th, th>>>(dev_a, dev_b, dev_c, N);
         hipEventRecord(stop, 0); 
         hipEventSynchronize(stop); 

         CUDA_CHECK_RETURN(hipGetLastError());
         hipEventElapsedTime(&elapsedTime, start, stop); 
         CUDA_CHECK_RETURN(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));
         fprintf(stderr, "%d: %.6f ms\n", N, elapsedTime);
         hipEventDestroy(start); 
         hipEventDestroy(stop);  
         free(c);
         hipFree(dev_a);
         hipFree(dev_b);
         hipFree(dev_c);
         cout << endl;    	
    }
    else if(argc == 1){
      for (int k = 1 << 0; k <= 1 << 10; k = k << 1) {
        fprintf(stderr, "Threads per block(%i):\n", k);
        for (int j = 10; j <= 23; j++) {
          hipEvent_t start, stop; 
          hipEventCreate(&start);
          hipEventCreate(&stop); 
          N = 1 << j;

          a = (int*)calloc(N, sizeof(int));
          b = (int*)calloc(N, sizeof(int));
          c = (int*)calloc(N, sizeof(int));
          init(a, b, c, N);
          CUDA_CHECK_RETURN(hipMalloc((void **)&dev_a, N * sizeof(int)));
          CUDA_CHECK_RETURN(hipMalloc((void **)&dev_b, N * sizeof(int)));
          CUDA_CHECK_RETURN(hipMalloc((void **)&dev_c, N * sizeof(int)));

          hipEventRecord(start, 0); 
          add<<<N / k, k>>>(dev_a, dev_b, dev_c, N);
          hipEventRecord(stop, 0); 
          hipEventSynchronize(stop); 

          CUDA_CHECK_RETURN(hipGetLastError());
          hipEventElapsedTime(&elapsedTime, start, stop);
          CUDA_CHECK_RETURN(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));
          fprintf(stderr, " %d: %.6f ms\n", N, elapsedTime);
          hipEventDestroy(start); 
          hipEventDestroy(stop);  
          free(c);
          hipFree(dev_a);
          hipFree(dev_b);
          hipFree(dev_c);
        }
        cout << endl;
      }
    }
    return 0;
}
