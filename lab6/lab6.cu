#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>


#define CUDA_CHECK_RETURN(value) {\
cudaError_t _m_cudaStat = value;\
if (_m_cudaStat != hipSuccess) {\
	fprintf(stderr, "Error %s at line %d in file %s\n",\
	hipGetErrorString(_m_cudaStat),__LINE__,__FILE__);\
	exit(1);\
}}


void allocTest(int size, bool hostToDevice, float &elapsed_time)
{
	hipEvent_t start, stop;
	int *a, *dev_a;

	CUDA_CHECK_RETURN(hipEventCreate(&start));
	CUDA_CHECK_RETURN(hipEventCreate(&stop));

	CUDA_CHECK_RETURN(hipHostAlloc((void**)&a, size * sizeof(*a), hipHostMallocDefault));
	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_a, size * sizeof(*dev_a)));

	for (int i = 0; i < size; i++)
    	a[i] = i;

	CUDA_CHECK_RETURN(hipEventRecord(start, 0));
	if (hostToDevice == true) {
		CUDA_CHECK_RETURN(hipMemcpy(dev_a, a, size * sizeof(*a), hipMemcpyHostToDevice));
	}
	else {
		CUDA_CHECK_RETURN(hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost));
	}

	CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
	CUDA_CHECK_RETURN(hipEventSynchronize(stop));
	CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsed_time, start, stop));

	CUDA_CHECK_RETURN(hipHostFree(a));
	CUDA_CHECK_RETURN(hipFree(dev_a));
	CUDA_CHECK_RETURN(hipEventDestroy(start));
	CUDA_CHECK_RETURN(hipEventDestroy(stop));
}

void mallocTest(int size, bool hostToDevice, float& elapsed_time)
{
	hipEvent_t start, stop;
	int *a, *dev_a;

	CUDA_CHECK_RETURN(hipEventCreate(&start));
	CUDA_CHECK_RETURN(hipEventCreate(&stop));

	a = (int*)malloc(size * sizeof(int));
	for (int i = 0; i < size; i++)
      	a[i] = i;
	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_a, size * sizeof(*dev_a)));

	CUDA_CHECK_RETURN(hipEventRecord(start, 0));
	if (hostToDevice == true) {
		CUDA_CHECK_RETURN(hipMemcpy(dev_a, a, size * sizeof(*a), hipMemcpyHostToDevice));
	}
	else {
		CUDA_CHECK_RETURN(hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost));
	}

	CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
	CUDA_CHECK_RETURN(hipEventSynchronize(stop));
	CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsed_time, start, stop));

	free(a);
	CUDA_CHECK_RETURN(hipFree(dev_a));
	CUDA_CHECK_RETURN(hipEventDestroy(start));
	CUDA_CHECK_RETURN(hipEventDestroy(stop));

}

int main(int argc, char const *argv[])
{
	hipDeviceProp_t deviceProp;
  	hipGetDeviceProperties(&deviceProp, 0);
  	printf("\nDevice:\t%s\n\n", deviceProp.name);
  	int size = 0;
  	if (argc == 2) {
    	size = 1 << atoi(argv[1]);
  	} else {
    	size = 1 << 13;
  	}
 	size *= size;
	float elapsed_time;
	printf("Size = %d\n\n", size);
	mallocTest(size, true, elapsed_time);
	printf("hipMalloc [host to device]: %.6f ms\n", elapsed_time);

	mallocTest(size, false, elapsed_time);
	printf("hipMalloc [device to host]: %.6f ms\n", elapsed_time);
	

	allocTest(size, true, elapsed_time);
	printf("hipHostAlloc [host to device]: %.6f ms\n", elapsed_time);
	
	allocTest(size, false, elapsed_time);
	printf("hipHostAlloc [device to host]: %.6f ms\n", elapsed_time);

	return 0;
}