#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#define N (2048*2048)
#define FULL_DATA_SIZE  (1024*1024*20)


bool check(float *host_a, float *host_b, float *host_c)
{
  for (int i = 0; i < FULL_DATA_SIZE; i++) {
    if (host_a[i] + host_b[i] - host_c[i] > 1e-5) {
      fprintf(stderr, "Wrong result!\n");
      return false;
    }
  }
  return true;
}

bool check(long long *host_a, long long *host_b, long long *host_c)
{
  for (long long i = 0; i < FULL_DATA_SIZE; i++) {
    if (host_a[i] * host_b[i] != host_c[i]) {
      fprintf(stderr, "Wrong result!\n");
      return false;
    }
  }
  return true;
}
__global__ void add(float*a, float *b, float *c)
{
  int idx = (blockDim.x * blockIdx.x) + threadIdx.x;
  if (idx < N) {
    c[idx] = a[idx] + b[idx];
  }
}

__global__ void mult(long long *a, long long *b, long long *c)
{
  int idx = (blockDim.x * blockIdx.x) + threadIdx.x;
  if (idx < N) {
    c[idx] = a[idx] * b[idx];
  }
}

void addVectors()
{
  float *host_a, *host_b, *host_c;
  float *dev_a0, *dev_b0, *dev_c0;
  float *dev_a1, *dev_b1, *dev_c1;

  hipMalloc((void**)&dev_a0, N * sizeof(float));
  hipMalloc((void**)&dev_b0, N * sizeof(float));
  hipMalloc((void**)&dev_c0, N * sizeof(float));
  hipMalloc((void**)&dev_a1, N * sizeof(float));
  hipMalloc((void**)&dev_b1, N * sizeof(float));
  hipMalloc((void**)&dev_c1, N * sizeof(float));

  hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(float), hipHostMallocDefault);
  hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(float), hipHostMallocDefault);
  hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(float), hipHostMallocDefault);

  srand(time(NULL));
  for (int i = 0; i < FULL_DATA_SIZE; i++) {
    host_a[i] = i + 1;
    host_b[i] = i + 1;
  }

  float elapsedTime;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipStream_t stream0, stream1;
  hipStreamCreate(&stream0);
  hipStreamCreate(&stream1);
  hipEventRecord(start, 0);

  for (int i = 0; i < FULL_DATA_SIZE; i += N * 2) {
    //printf("i = %d\n", i);
    hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(float), hipMemcpyHostToDevice, stream0);
    hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(float), hipMemcpyHostToDevice, stream1);

    hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(float), hipMemcpyHostToDevice, stream0);
    hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(float), hipMemcpyHostToDevice, stream1);

    add << <N / 256, 256, 0, stream0 >> >(dev_a0, dev_b0, dev_c0);
    add << <N / 256, 256, 0, stream1 >> >(dev_a1, dev_b1, dev_c1);

    hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(float), hipMemcpyDeviceToHost, stream0);
    hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(float), hipMemcpyDeviceToHost, stream1);
  }

  hipStreamSynchronize(stream0);
  hipStreamSynchronize(stream1);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);

  if (!check(host_a, host_b, host_c))
    printf("Something goes wrong!\n");

  printf("Elapsed time: %3.1f ms\n", elapsedTime);

  hipHostFree(host_a);
  hipHostFree(host_b);
  hipHostFree(host_c);

  hipFree(dev_a0);
  hipFree(dev_b0);
  hipFree(dev_c0);
  hipFree(dev_a1);
  hipFree(dev_b1);
  hipFree(dev_c1);

  hipStreamDestroy(stream0);
  hipStreamDestroy(stream1);

}

void multVectors()
{
  long long *host_a, *host_b, *host_c;
  long long *dev_a0, *dev_b0, *dev_c0;
  long long *dev_a1, *dev_b1, *dev_c1;

  hipMalloc((void**)&dev_a0, N * sizeof(long long));
  hipMalloc((void**)&dev_b0, N * sizeof(long long));
  hipMalloc((void**)&dev_c0, N * sizeof(long long));
  hipMalloc((void**)&dev_a1, N * sizeof(long long));
  hipMalloc((void**)&dev_b1, N * sizeof(long long));
  hipMalloc((void**)&dev_c1, N * sizeof(long long));

  hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(long long), hipHostMallocDefault);
  hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(long long), hipHostMallocDefault);
  hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(long long), hipHostMallocDefault);

  for (long long i = 0; i < FULL_DATA_SIZE; i++) {
    host_a[i] = i + 1;
    host_b[i] = i + 1;
  }

  float elapsedTime;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipStream_t stream0, stream1;
  hipStreamCreate(&stream0);
  hipStreamCreate(&stream1);
  hipEventRecord(start, 0);

  for (int i = 0; i < FULL_DATA_SIZE; i += N * 2) {
    //printf("i = %d\n", i);
    hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(long long), hipMemcpyHostToDevice, stream0);
    hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(long long), hipMemcpyHostToDevice, stream1);

    hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(long long), hipMemcpyHostToDevice, stream0);
    hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(long long), hipMemcpyHostToDevice, stream1);

    mult << <N / 256, 256, 0, stream0 >> >(dev_a0, dev_b0, dev_c0);
    mult << <N / 256, 256, 0, stream1 >> >(dev_a1, dev_b1, dev_c1);

    hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(long long), hipMemcpyDeviceToHost, stream0);
    hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(long long), hipMemcpyDeviceToHost, stream1);
  }

  hipStreamSynchronize(stream0);
  hipStreamSynchronize(stream1);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);

  if (!check(host_a, host_b, host_c))
    printf("Something goes wrong!\n");

  printf("Elapsed time: %3.1f ms\n", elapsedTime);

  hipHostFree(host_a);
  hipHostFree(host_b);
  hipHostFree(host_c);

  hipFree(dev_a0);
  hipFree(dev_b0);
  hipFree(dev_c0);
  hipFree(dev_a1);
  hipFree(dev_b1);
  hipFree(dev_c1);

  hipStreamDestroy(stream0);
  hipStreamDestroy(stream1);
}

int main(int argc, char const *argv[])
{
  hipDeviceProp_t prop;
  int whichDevice;

  hipGetDevice(&whichDevice);
  hipGetDeviceProperties(&prop, whichDevice);
  if (!prop.deviceOverlap) {
    printf("Device does not support overlapping\n");
    return 0;
  }
  printf("FULL_DATA_SIZE = %d\n", FULL_DATA_SIZE);
  printf("N = %d\n", N);
  printf("Add vectors: \n");
  addVectors();

  printf("Mult vectors: \n");
  multVectors();
  return 0;
}