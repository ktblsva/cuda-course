#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#define M_PI 3.14159265358979323846
#define COEF 48
#define VERTCOUNT COEF *COEF * 2
#define RADIUS 160.0f
#define FGSIZE 320
#define FGSHIFT FGSIZE / 2
#define IMIN(A, B) (A < B ? A : B)
#define THREADSPERBLOCK 256
#define BLOCKSPERGRID                                                          \
  IMIN(32, (VERTCOUNT + THREADSPERBLOCK - 1) / THREADSPERBLOCK)
#define CUDA_CHECK_RETURN(value)                                               \
  {                                                                            \
    hipError_t _m_cudaStat = value;                                           \
    if (_m_cudaStat != hipSuccess) {                                          \
      fprintf(stderr, "Error %s at line %d in file %s\n",                      \
              hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);            \
      exit(1);                                                                 \
    }                                                                          \
  }
typedef float (*ptr_f)(float, float, float);

struct Vertex {
  float x, y, z;
};

__constant__ Vertex vert[VERTCOUNT];

texture<float, 3, hipReadModeElementType> df_tex;
hipArray *df_Array = 0;

float func(float x, float y, float z) {
  return (0.5 * sqrtf(15.0 / M_PI)) * (0.5 * sqrtf(15.0 / M_PI)) * z * z * y *
         y * sqrtf(1.0f - z * z / RADIUS / RADIUS) / RADIUS / RADIUS / RADIUS /
         RADIUS;
}

// Проверка суммы по функции в декартовых координатах
float check(Vertex *v, ptr_f f) {
  float sum = 0.0f;

  for (int i = 0; i < VERTCOUNT; i++) {
    sum += f(v[i].x, v[i].y, v[i].z);
  }
  return sum;
}

void calc_f(float *arr_f, int x_size, int y_size, int z_size, ptr_f f) {
  for (int x = 0; x < x_size; x++)
    for (int y = 0; y < y_size; y++)
      for (int z = 0; z < z_size; z++)
        arr_f[z_size * (x * y_size + y) + z] =
            f(x - FGSHIFT, y - FGSHIFT, z - FGSHIFT);
}

void init_vertices(Vertex *vertex_dev) {
  Vertex *temp_vert = (Vertex *)malloc(sizeof(Vertex) * VERTCOUNT);
  int i = 0;
  for (int iphi = 0; iphi < 2 * COEF; iphi++) {
    for (int ipsi = 0; ipsi < COEF; ipsi++, i++) {
      float phi = iphi * M_PI / COEF;
      float psi = ipsi * M_PI / COEF;
      temp_vert[i].x = RADIUS * sinf(psi) * cosf(phi);
      temp_vert[i].y = RADIUS * sinf(psi) * sinf(phi);
      temp_vert[i].z = RADIUS * cosf(psi);
    }
  }
  printf("Проверка суммы = %f\n",
         check(temp_vert, &func) * M_PI * M_PI / COEF / COEF);
  // Функция для копирования данных в текстурную память
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(vert), temp_vert, sizeof(Vertex) * VERTCOUNT, 0, hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(vertex_dev, temp_vert, sizeof(Vertex) * VERTCOUNT, hipMemcpyHostToDevice));
  free(temp_vert);
}

void init_texture(float *df_h) {
  const hipExtent volumeSize = make_hipExtent(FGSIZE, FGSIZE, FGSIZE);
  // Формат дескриптора канала 
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

  hipMalloc3DArray(&df_Array, &channelDesc, volumeSize);
  hipMemcpy3DParms cpyParams = {0};
  // Адрес исходной памяти
  cpyParams.srcPtr =
      make_hipPitchedPtr((void *)df_h, volumeSize.width * sizeof(float),
                          volumeSize.width, volumeSize.height);
  // df_h - Указатель на выделенную память
  // volumeSize.width * sizeof(float) - шаг выделенной памяти в байтах
  // volumeSize.width - логическая ширина(высота) размещения в элементах
  cpyParams.dstArray = df_Array;
  // Запрошенный размер экземпляра памяти
  cpyParams.extent = volumeSize;
  // Тип копирования
  cpyParams.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&cpyParams);

  df_tex.normalized = false; // Указывает, нормализовано ли чтение текстуры или нет
  df_tex.filterMode = hipFilterModeLinear; // hipFilterModePoint | hipFilterModeLinear
  // Режим текстурной адресации для 3-х измерений
  df_tex.addressMode[0] = hipAddressModeClamp; // Clamp зацикливает?
  df_tex.addressMode[1] = hipAddressModeClamp;
  df_tex.addressMode[2] = hipAddressModeClamp;
  // Привязывает массив к текстуре
  hipBindTextureToArray(df_tex, df_Array, channelDesc);
}

void release_texture() {
  hipUnbindTexture(df_tex);
  hipFreeArray(df_Array);
}

__global__ void kernel(float *a) {
  // Использование разделяемой памяти для кеширования фильтрованных значений функции
  __shared__ float cache[THREADSPERBLOCK];
  // Индекс потока
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int cacheIndex = threadIdx.x;
  // Получаем координаты вершин в которых нужно посчитать значение функции + сдвигаем в центр
  float x, y, z;
  x = vert[tid].x + FGSHIFT + 0.5f;
  y = vert[tid].y + FGSHIFT + 0.5f;
  z = vert[tid].z + FGSHIFT + 0.5f;

  cache[cacheIndex] = tex3D(df_tex, z, y, x);

  __syncthreads();

  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (cacheIndex < s)
      cache[cacheIndex] += cache[cacheIndex + s];
    __syncthreads();
  }
  if (cacheIndex == 0)
    a[blockIdx.x] = cache[0];
}

__device__ float getDistance(Vertex a, Vertex b) {
  return sqrtf((a.x - b.x) * (a.x - b.x) + (a.y - b.y) * (a.y - b.y) + (a.z - b.z) * (a.z - b.z));
}

__device__ float interpolateStep(float *arr_f, float z, float y, float x) {

  int gx = x;
  int gy = y;
  int gz = z;

  //за пределы сетки
  if (gx + 1 >= FGSIZE || gy + 1 >= FGSIZE || gz + 1 >= FGSIZE)
    return 0.0;

  float fgx = float(gx);
  float fgy = float(gy);
  float fgz = float(gz);

  //углы куба
  Vertex angle[8] = {{fgx, fgy, fgz},      {fgx + 1, fgy, fgz},      {fgx, fgy + 1, fgz},      {fgx + 1, fgy + 1, fgz},
                      {fgx, fgy, fgz + 1},  {fgx + 1, fgy, fgz + 1},  {fgx, fgy + 1, fgz + 1},  {fgx + 1, fgy + 1, fgz + 1}};
                                   
  // arr_f[z_size * (x * y_size + y) + z]
  float value = arr_f[FGSIZE * (gx * FGSIZE + gy) + gz];
  Vertex vrt {angle[0].x, angle[0].y, angle[0].z};
  Vertex v {x, y, z};
  float distance = getDistance(vrt, v);
  float tmp = 0;

  //минимальное расстояние к точке
  for (int i = 1; i < 8; i++) {
    Vertex vrt1;
    vrt1.x = angle[i].x;
    vrt1.y = angle[i].y;
    vrt1.z = angle[i].z;
    tmp = getDistance(vrt1, v);
    if (tmp < distance) {
      distance = tmp;
      value = arr_f[FGSIZE * (int(angle[i].x) * FGSIZE + int(angle[i].y)) +
                    int(angle[i].z)];
    }
  }

  return value;
}

__global__ void proximalInterpolation(float *a, float *arr, Vertex *v)
{
  __shared__ float cache[THREADSPERBLOCK];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int cacheIndex = threadIdx.x;
  float x, y, z;
  x = vert[tid].x + FGSHIFT + 0.5f;
  y = vert[tid].y + FGSHIFT + 0.5f;
  z = vert[tid].z + FGSHIFT + 0.5f;

  cache[cacheIndex] = interpolateStep(arr, z, y, x);

  __syncthreads();

  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (cacheIndex < s)
      cache[cacheIndex] += cache[cacheIndex + s];
    __syncthreads();
  }
  if (cacheIndex == 0)
    a[blockIdx.x] = cache[0];
}

__device__ float interpolate1D(float a, float b, float x) {
  return a * (1 - x) + b * x;
}

__device__ float interpolate2D(float a1, float b1, float a2, float b2, float x, float y) {

  float v1 = interpolate1D(a1, b1, x);
  float v2 = interpolate1D(a2, b2, x);

  return interpolate1D(v1, v2, y);
}

__device__ float interpolate3D(float *arr_f, float z, float y, float x) {

  int gx = x;
  int gy = y;
  int gz = z;
  float tx = x - (float)gx;
  float ty = z - (float)gz;
  float tz = z - (float)gz;

  if (gx + 1 >= FGSIZE || gy + 1 >= FGSIZE || gz + 1 >= FGSIZE)
    return 0.0f;

  float c000 = arr_f[FGSIZE * (gx * FGSIZE + gy) + gz];
  float c001 = arr_f[FGSIZE * ((gx + 1) * FGSIZE + gy) + gz];
  float c010 = arr_f[FGSIZE * (gx * FGSIZE + (gy + 1)) + gz];
  float c011 = arr_f[FGSIZE * ((gx + 1) * FGSIZE + (gy + 1)) + gz];

  float c100 = arr_f[FGSIZE * (gx * FGSIZE + gy) + (gz + 1)];
  float c101 = arr_f[FGSIZE * ((gx + 1) * FGSIZE + gy) + (gz + 1)];
  float c110 = arr_f[FGSIZE * (gx * FGSIZE + (gy + 1)) + (gz + 1)];
  float c111 = arr_f[FGSIZE * ((gx + 1) * FGSIZE + (gy + 1)) + (gz + 1)];

  float e = interpolate2D(c000, c001, c010, c011, tx, ty);
  float f = interpolate2D(c100, c101, c110, c111, tx, ty);

  return interpolate1D(e, f, tz);
}

__global__ void trilinearInterpolation(float *a, float *arr, Vertex *v) {
  __shared__ float cache[THREADSPERBLOCK];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int cacheIndex = threadIdx.x;
  float x = v[tid].x + FGSHIFT;
  float y = v[tid].y + FGSHIFT;
  float z = v[tid].z + FGSHIFT;

  cache[cacheIndex] = interpolate3D(arr, z, y, x);
  __syncthreads();

  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (cacheIndex < s)
      cache[cacheIndex] += cache[cacheIndex + s];
    __syncthreads();
  }
  if (cacheIndex == 0)
    a[blockIdx.x] = cache[0];
}

int main(int argc, char *argv[]) {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  printf("\nDevice:\t%s\n\n", deviceProp.name);

  Vertex *vert_dev;
  float elapsedTime;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float *arr = (float *)malloc(sizeof(float) * FGSIZE * FGSIZE * FGSIZE);
  float *sum = (float *)malloc(sizeof(float) * BLOCKSPERGRID);
  float *sum_dev, *arr_dev;

  CUDA_CHECK_RETURN(hipMalloc((void **)&sum_dev, sizeof(float) * BLOCKSPERGRID));
  CUDA_CHECK_RETURN(hipMalloc((void **)&arr_dev, sizeof(float) * FGSIZE * FGSIZE * FGSIZE));
  CUDA_CHECK_RETURN(hipMalloc((void **)&vert_dev, sizeof(Vertex) * VERTCOUNT));

  init_vertices(vert_dev);
  calc_f(arr, FGSIZE, FGSIZE, FGSIZE, &func);
  init_texture(arr);

  CUDA_CHECK_RETURN(hipMemcpy(arr_dev, arr, sizeof(float) * FGSIZE * FGSIZE * FGSIZE,
             hipMemcpyHostToDevice));
  /* Texture Kernel */
  hipEventRecord(start, 0);
  kernel<<<BLOCKSPERGRID, THREADSPERBLOCK>>>(sum_dev);
  CUDA_CHECK_RETURN(hipGetLastError());
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  CUDA_CHECK_RETURN(hipMemcpy(sum, sum_dev, sizeof(float) * BLOCKSPERGRID, hipMemcpyDeviceToHost));

  float s = 0.0f;
  for (int i = 0; i < BLOCKSPERGRID; i++) {
    s += sum[i];
  }
  hipEventElapsedTime(&elapsedTime, start, stop);
  fprintf(stderr, "TextureSum = %f\n",s * M_PI * M_PI / COEF / COEF);

 /* Proximal Interpolation */
  hipEventRecord(start, 0);
  proximalInterpolation<<<BLOCKSPERGRID, THREADSPERBLOCK>>>(sum_dev, arr_dev, vert_dev);
  CUDA_CHECK_RETURN(hipGetLastError());
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  CUDA_CHECK_RETURN(hipMemcpy(sum, sum_dev, sizeof(float) * BLOCKSPERGRID, hipMemcpyDeviceToHost));

  s = 0.0f;
  for (int i = 0; i < BLOCKSPERGRID; i++) {
    s += sum[i];
  }
  hipEventElapsedTime(&elapsedTime, start, stop);
  fprintf(stderr, "ProximalInterpolationSum = %f\n", s * M_PI * M_PI / COEF / COEF);

 /* Trilinear Kernel */
  hipEventRecord(start, 0);
  trilinearInterpolation<<<BLOCKSPERGRID, THREADSPERBLOCK>>>(sum_dev, arr_dev, vert_dev);
  CUDA_CHECK_RETURN(hipGetLastError());
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  CUDA_CHECK_RETURN(hipMemcpy(sum, sum_dev, sizeof(float) * BLOCKSPERGRID, hipMemcpyDeviceToHost));

  s = 0.0f;
  for (int i = 0; i < BLOCKSPERGRID; i++) {
    s += sum[i];
  }
  hipEventElapsedTime(&elapsedTime, start, stop);
  fprintf(stderr, "TrilinearInterpolationSum = %f\n", s * M_PI * M_PI / COEF / COEF);
  hipFree(sum);
  release_texture();
  free(arr);
  return 0;
}