#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <stdio.h>
#include <fstream>
#include <string.h>

using namespace std;

#define step 256
#define N 1024
#define Nt 20
#define ALPHA 0.2
#define T 2
struct functor
{
    const float koef;
    functor(float _koef) : koef(_koef){}
    __host__ __device__ float operator()(float x, float y)
    {
        return x + koef * (y - x);
    }
};


__global__ void kernel(float *f, float *res)
{
    int cur = threadIdx.x + blockDim.x * blockIdx.x;
    int prev = cur - 1;
    if(prev == -1)
    {
        res[cur] = f[cur];
    }else
    {
        res[cur] = f[cur] + ALPHA * T * (f[prev] - f[cur]);
    }
}

int data(int curr)
{
	// if(curr < 4 || curr >= 8 && curr < 12) return 0;
	// else return 1;
	return (curr / step) % 2 == 1;
}

int main()
{
	float funct[N * Nt];
	float funcData[N * Nt];
	float *temp;

	hipEvent_t start, stop;
	float time;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	for(int i = 0; i < N; i++) {
       for(int j = 0; j < Nt; j++) {
                funcData[i + j * Nt] = 0;
        }
    }
     for(int i = 0; i < N; i++) {
        funcData[i + 0 * Nt] = data(i);
        printf("%0.2f ", funcData[i + 0 * Nt]);
    }
	printf("\n\n");
	hipMalloc((void **)&temp, sizeof(float) * N * Nt);
	hipMemcpy(temp, funcData, sizeof(float) * N * Nt, hipMemcpyHostToDevice);

	hipEventSynchronize(start);
	hipEventRecord(start, 0);
	for (int i = 0; i < Nt; i++) {
		kernel <<< 1, N >>> (temp + (i * N), temp + ((i + 1)* N));
		hipDeviceSynchronize();
	}
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipMemcpy(funct, temp, N * Nt * sizeof(float), hipMemcpyDeviceToHost);
	printf("cuda: %f ms\n", time);
	ofstream of("x.txt");

	for(int i = 0; i < Nt; i++) {
		printf("f[%d] ", i);
	}
	for(int i = 0; i < N; i++) {
		of << i << endl;
	}
	printf("\n");
 	for(int i = 0; i < N; i++) {
       for(int j = 0; j < Nt; j++) {
            printf("%0.2f ", funct[i + j * Nt]);
        }
        printf("\n");
    }
    for(int i = 0; i < Nt; i++)
    {
    	string num("y" + to_string(i) + ".txt");
    	ofstream y(num);
    	for(int j = 0; j < N; j++)
    	{
    		y << funct [j + i * N]<< endl;
    	}
    }

	thrust::host_vector<float> vect(N * 10);

	for (int i = 0; i < N; i++)
	{
		vect[i] = funcData[i];
	}
	thrust::device_vector<float> x(N * 10);
	thrust::copy(vect.begin(), vect.end(),x.begin());
	hipEventSynchronize(start);
	hipEventRecord(start, 0);
	functor func(ALPHA * T);
	for(int j = 0; j < Nt; j++){
        thrust::transform(x.begin()+(j*N)+1, x.begin()+((j+1)*N), x.begin()+(j*N), x.begin() +((j+1)*N)+1, func);
    }
    thrust::copy(x.begin(),x.end(),vect.begin());
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	std::cout << std::endl;
	printf("thrust: %f ms\n", time);
	for(int i = 0; i < Nt; i++)
	{
		printf("f[%d] ", i);
	}
	printf("\n");
 	for(int i = 0; i < N; i++){
       for(int j = 0; j < Nt; j++){
                printf("%0.2f ", vect[i + j * Nt]);
        }
        printf("\n");
    }
    system("python3 1.py");
    return 0;
}