#include "hip/hip_runtime.h"
#include ""
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <string.h>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include <cctype>
#include <list>
#include <stdlib.h>
#include <ctime>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>


#include <hipblas.h>
#include <hipblas.h>

#define CUDA_CHECK_RETURN(value)                                               \
  {                                                                            \
    hipError_t _m_cudaStat = value;                                           \
    if (_m_cudaStat != hipSuccess) {                                          \
      fprintf(stderr, "Error %s at line %d in file %s\n",                      \
              hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);            \
      exit(1);                                                                 \
    }                                                                          \
  }                                                              \


#define NX 64
#define BATCH 1
#define pi 3.141592
#define SZ (1<<23)
#define ALPHA 2.0f

using namespace std;


__global__ void cusaxpy(int n, float a, float *x, float *y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) y[i] = a*x[i] + y[i];
}


struct functor {
        const float koef;
        functor(float _koef) : koef(_koef) {}
        __host__ __device__ float operator()(float x, float y) { return koef * x + y; }
};

void saxpy(float _koef, thrust::device_vector<float> &x, thrust::device_vector<float> &y)
{
        functor func(_koef);
        thrust::transform(x.begin(), x.end(), y.begin(), y.begin(), func);

}


int thrustFunc(float* a, int N) {
	hipEvent_t start, stop;
	float time;
	printf("here!\n");
	thrust::host_vector<float> h1(N);
       	thrust::host_vector<float> h2(N);
        float alpha = ALPHA;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        for(int k = 0; k < N; k++){
                h1[k] = a[k];
                h2[k] = a[k];
                	//printf("%g ", h1[k]);
        }
        //printf("\n");
        thrust::device_vector<float> gpumem1 = h1;
        thrust::device_vector<float> gpumem2 = h2;
        printf("after gpu!\n");
        hipEventRecord(start, 0);
        saxpy(alpha, gpumem1, gpumem2);
  	hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        printf("after saxpy!\n");
        h1 = gpumem1;
        h2 = gpumem2;
        for(int k = 0; k < N; k++) {
        	if(h2[k] == h1[k]*ALPHA + h1[k]) {
        		continue;
        	} else {
        		printf("Thrust: wrong answer!\n");
        		return -1;
        	}
        	printf("%g\t %g\n", h1[k], h2[k]);
        }
        printf("Thrust: correct answer!\n");
        
        printf("Thrust time: %g ms \n", time);
        return 0;
}

int cublasFunc(float* a, int N) {
	hipEvent_t start, stop;
	float time;
	hipblasHandle_t handle;
        hipblasCreate(&handle);
        float *res = new float[N];
        float *dev_x, *dev_y;
        hipMalloc(&dev_x, N * sizeof(float));
	hipMalloc(&dev_y, N * sizeof(float));
	hipEventCreate(&start);
        hipEventCreate(&stop);
	cublasInit();
	hipblasSetVector(N, sizeof(a[0]), a, 1, dev_x, 1);
        hipblasSetVector(N, sizeof(a[0]), a, 1, dev_y, 1);
        float alpha = ALPHA;
	hipEventRecord(start, 0);
	hipblasSaxpy(handle, N, &alpha, dev_x, 1, dev_y, 1);
	hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipblasGetVector(N, sizeof(res[0]), dev_y, 1, res, 1);
	cublasShutdown();
	for(int k = 0; k < N; k++) {
        	if(res[k] == a[k]*ALPHA + a[k]) {
        		continue;
        	} else {
        		printf("cuBLAS: wrong answer!\n");
        		return -1;
        	}
        	printf("%g\t %g\n", a[k], res[k]);
        }
        printf("cuBLAS: correct answer!\n");
        
        printf("cuBLAS time: %g ms \n", time);
	hipblasDestroy(handle);
	hipFree(dev_x);
	hipFree(dev_y);
	hipHostFree(res);
	hipDeviceReset();
	return 0;
}

int cudaFunc(float* a, int N) {
	hipEvent_t start, stop;
	float time;
	float alpha = ALPHA;        
        hipEventCreate(&start); 
        hipEventCreate(&stop);
	float *x, *y, *d_x, *d_y;
	x = (float*)malloc(N*sizeof(float));
	y = (float*)malloc(N*sizeof(float));
	hipMalloc(&d_x, N*sizeof(float));
	hipMalloc(&d_y, N*sizeof(float));
	for (int k = 0; k < N; k++) {
		x[k] = k;
		y[k] = k;
	}
	hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
	hipEventRecord(start,0);
	cusaxpy<<<(N)/256, 256>>>(N, alpha, d_x, d_y);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
	hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
   
	for(int k = 0; k < N; k++) {
        	if(y[k] == a[k]*ALPHA + a[k]) {
        		continue;
        	} else {
        		printf("cuda C: wrong answer!\n");
        		return -1;
        	}
        	//printf("%g\t %g\n", a[k], y[k]);
        }
        hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);
        printf("cuda C: correct answer!\n");
        printf("cuda C time: %g ms \n", time);
        return 0;
}


int main() {
        srand(time(nullptr));
	float *a = (float*)calloc(SZ, sizeof(float));
    
	for (int i = 0; i < SZ; i++) {
		a[i] = i;
	}

	ofstream os("N.dat");
	ofstream th("threads.dat");
	ofstream os1("dataThrust.dat");
	ofstream os2("dataCublas.dat");
	ofstream os3("dataCuda.dat");

    	if(!os1.is_open()){
        	cout << "Error" << endl;
        	return -1;
   	}
    	if(!os2.is_open()){
                cout << "Error" << endl;
                return -1;
        }
	if(!os3.is_open()){
                cout << "Error" << endl;
                return -1;
        }

	int i = 256;
       	for (int j = 10; j <= 23; j++) {
		//int N = 1 << j; 
		int N = 2048;
		printf("N = %d\n", N);
		if (thrustFunc(a, N) != 0) {
			return -1;
		}
		if (cublasFunc(a, N) != 0) {
			return -1;
		}
		cudaFunc(a, N);
	
	}

	os1.close();
	os2.close();
	os3.close();
	return 0;
}