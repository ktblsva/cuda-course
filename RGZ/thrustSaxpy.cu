#include ""
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <string.h>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include <cctype>
#include <list>
#include <stdlib.h>
#include <ctime>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#define ALPHA 2.0f
#define SZ (1<<23)

using namespace std;

struct functor {
        const float koef;
        functor(float _koef) : koef(_koef) {}
        __host__ __device__ float operator()(float x, float y) { return koef * x + y; }
};

void saxpy(float _koef, thrust::device_vector<float> &x, thrust::device_vector<float> &y)
{
        functor func(_koef);
        thrust::transform(x.begin(), x.end(), y.begin(), y.begin(), func);

}

int main(void)
{
	float *a = (float*)calloc(SZ, sizeof(float));
    ofstream os("N.dat");
	ofstream os1("dataThrust.dat");

	for (int i = 0; i < SZ; i++) {
		a[i] = i;
	}
	for(int i = 10; i <= 23; i++) {
		int N = 1 << i;
		printf("N = %d\n", N);
		os << N << endl;
		hipEvent_t start, stop;
		float time;
		thrust::host_vector<float> h1(N);
	    thrust::host_vector<float> h2(N);
	    float alpha = ALPHA;
	    hipEventCreate(&start);
	    hipEventCreate(&stop);
	    for(int k = 0; k < N; k++){
	            h1[k] = a[k];
	            h2[k] = a[k];
	    }
	     
	    thrust::device_vector<float> gpumem1 = h1;
	    thrust::device_vector<float> gpumem2 = h2;
	    hipEventRecord(start, 0);
	    saxpy(alpha, gpumem1, gpumem2);
	  	hipEventRecord(stop, 0);
	    hipEventSynchronize(stop);
	    hipEventElapsedTime(&time, start, stop);
	    h1 = gpumem1;
	    h2 = gpumem2;
	    for(int k = 0; k < N; k++) {
	        if(h2[k] == h1[k]*ALPHA + h1[k]) {
	        		continue;
	        } else {
	        		printf("Thrust: wrong answer!\n");
	        		return -1;
	        }
	        printf("%g\t %g\n", h1[k], h2[k]);
	    }
	    printf("Thrust: correct answer!\n");
	    printf("Thrust time: %g ms \n", time);
	    os1 << time << endl;
        
    }
}
