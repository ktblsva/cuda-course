#include ""
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <string.h>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include <cctype>
#include <list>
#include <stdlib.h>
#include <ctime>
#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <hipblas.h>
#define NX 64
#define BATCH 1
#define pi 3.141592
#define SZ (1<<23)
#define ALPHA 2.0f
using namespace std;

int main() {
	float *a = (float*)calloc(SZ, sizeof(float));
	ofstream os1("dataCublas.dat");

	for (int i = 0; i < SZ; i++) {
		a[i] = i;
	}
	for(int i = 10; i <=23; i++) {
		int N = 1 << i;
		printf("N = %d\n", N);
		hipEvent_t start, stop;
		float time;
		hipblasHandle_t handle;
	    hipblasCreate(&handle);
	    float *res = new float[N];
	    float *dev_x, *dev_y;
	    hipMalloc(&dev_x, N * sizeof(float));
		hipMalloc(&dev_y, N * sizeof(float));
		hipEventCreate(&start);
	    hipEventCreate(&stop);
		cublasInit();
		hipblasSetVector(N, sizeof(a[0]), a, 1, dev_x, 1);
	    hipblasSetVector(N, sizeof(a[0]), a, 1, dev_y, 1);
	    float alpha = ALPHA;
		hipEventRecord(start, 0);
		hipblasSaxpy(handle, N, &alpha, dev_x, 1, dev_y, 1);
		hipEventRecord(stop, 0);
	    hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);

		hipblasGetVector(N, sizeof(res[0]), dev_y, 1, res, 1);
		cublasShutdown();
		for(int k = 0; k < N; k++) {
	        	if(res[k] == a[k]*ALPHA + a[k]) {
	        		continue;
	        	} else {
	        		printf("cuBLAS: wrong answer!\n");
	        		return -1;
	        	}
	        	printf("%g\t %g\n", a[k], res[k]);
	        }
	    printf("cuBLAS: correct answer!\n");
	        
	    printf("cuBLAS time: %g ms \n", time);
	    os1 << time << endl;
		hipblasDestroy(handle);
		hipFree(dev_x);
		hipFree(dev_y);
		hipHostFree(res);
		hipDeviceReset();
	}
}