
#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void MyKernel(int *array, int arrayCount) 
{ 
  int idx = threadIdx.x + blockIdx.x * blockDim.x; 
  if (idx < arrayCount) 
  { 
    array[idx] *= array[idx]; 
  } 
} 

void launchMyKernel(int *array, int arrayCount) 
{ 
  int blockSize;   // The launch configurator returned block size 
  int minGridSize; // The minimum grid size needed to achieve the 
                   // maximum occupancy for a full device launch 
  int gridSize;    // The actual grid size needed, based on input size 

  hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, 
                                      MyKernel, 0, 0); 
  // Round up according to array size 
  gridSize = (arrayCount + blockSize - 1) / blockSize; 

  MyKernel<<< gridSize, blockSize >>>(array, arrayCount); 

  hipDeviceSynchronize(); 

  // calculate theoretical occupancy
  int maxActiveBlocks;
  hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, 
                                                 MyKernel, blockSize, 
                                                 0);

  int device;
  hipDeviceProp_t props;
  hipGetDevice(&device);
  hipGetDeviceProperties(&props, device);

  float occupancy = (maxActiveBlocks * blockSize / props.warpSize) / 
                    (float)(props.maxThreadsPerMultiProcessor / 
                            props.warpSize);

  printf("Max active block size: %d.\n", 
         maxActiveBlocks);
  printf("Launched blocks of size %d. Theoretical occupancy: %f\n", 
         blockSize, occupancy);
}

int main(int argc,char* argv[])
{
    int N = 0;
    int *c;
    N = atoi(argv[1]);
    c = (int*)calloc(N, sizeof(int));
    for(int k = 0; k < N; k++)
    {
      c[k] = k;
    }
    launchMyKernel(c, N);
    return 0;
}