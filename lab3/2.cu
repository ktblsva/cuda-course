#include <hip/hip_runtime.h>

#include <iomanip>
#include <iostream>
#include <malloc.h>
#include <stdio.h>

using namespace std;

#define CUDA_CHECK_RETURN(value)                                               \
  {                                                                            \
    hipError_t _m_cudaStat = value;                                           \
    if (_m_cudaStat != hipSuccess) {                                          \
      fprintf(stderr, "Error %s at line %d in file %s\n",                      \
              hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);            \
      exit(1);                                                                 \
    }                                                                          \
  }

__global__ void init(int *c, int N) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
  if(i >= N) return;
  c[i] = 0;
}


int main(int argc,char* argv[]) 
{ 
    // char dev;
    // cudaSetDevice(dev); 
    // cudaDeviceProp deviceProp; 
    // cudaGetDeviceProperties(&deviceProp, dev);
    // printf("  Total amount of constant memory:  %lu bytes\n", deviceProp.totalConstMem); 
    // printf("  Total amount of shared memory per block: %lu bytes\n", deviceProp.sharedMemPerBlock);
    // printf("  Total number of registers available per block: %d\n", deviceProp.regsPerBlock); 
    // printf("  Warp size: %d\n", deviceProp.warpSize); 
    // printf("  Maximum number of threads per multiprocessor:  %d\n", deviceProp.maxThreadsPerMultiProcessor); 
    // printf("  Maximum number of threads per block:  %d\n", deviceProp.maxThreadsPerBlock);
    float elapsedTime;
    int N = 0;
    int *dev_c, *c;
    N = atoi(argv[1]);
    int th = atoi(argv[2]);
    hipEvent_t start, stop;          
    hipEventCreate(&start); 
    hipEventCreate(&stop);
    fprintf(stderr, "%d blocks\n", N / th);
    c = (int*)calloc(N, sizeof(int));
    CUDA_CHECK_RETURN(hipMalloc((void **)&dev_c, N * sizeof(int)));

    hipEventRecord(start, 0); 
    init<<<N / th, th>>>(dev_c, N);
    hipEventRecord(stop, 0); 
    hipEventSynchronize(stop); 

    CUDA_CHECK_RETURN(hipGetLastError());
    hipEventElapsedTime(&elapsedTime, start, stop); 
    CUDA_CHECK_RETURN(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));
    fprintf(stderr, "%d: %.6f ms\n", N, elapsedTime);
    hipEventDestroy(start); 
    hipEventDestroy(stop);  
    free(c);
    hipFree(dev_c);
    cout << endl;    	
    return 0;
}
